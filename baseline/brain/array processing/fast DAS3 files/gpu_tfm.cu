
#include <hip/hip_runtime.h>
__global__ void tfm_near_norm(float* real_result,float* imag_result,const int n,const int combs, const float* real_exp,const float* img_exp,const int* transmit,const int* receive,const int* lookup_ind, const int tot_pix, const int grid_x, const int grid_y, const int grid_z, const float* lookup_amp, const float* tt_weight){

	// get pixel's coordinates
    int pix = blockIdx.x*blockDim.x+threadIdx.x;
     
    if (pix<tot_pix){
            //local variable
            float tot_real = 0, tot_imag = 0;

            for(int ii = 0; ii < combs; ii++){
                float real = 0;
                float imag = 0;
                int tx = transmit[ii]-1;
                int rx = receive[ii]-1;
                int t_ind = (tx*grid_x*grid_y*grid_z)+pix;
                int r_ind = (rx*grid_x*grid_y*grid_z)+pix;

                int index = lookup_ind[t_ind] + lookup_ind[r_ind] -1; 
                float amp_corr = lookup_amp[t_ind]*lookup_amp[r_ind]*tt_weight[ii];
                if(index<0){
                    }
                else if(index>n){
                    }
                else
                    {
                    int set_val = ii*(n)+index;
                    real = real_exp[set_val];
                    real = real*amp_corr;
                    imag = img_exp[set_val];
                    imag = imag*amp_corr;
                    }
                // sum each val
                tot_real += real;
                tot_imag += imag;
					
            }
		
            // store the final value for the pixel
            //result[pix] = sqrt(tot_real*tot_real + tot_imag*tot_imag);
            real_result[pix] = tot_real;
            imag_result[pix] = tot_imag;
    }
}

__global__ void tfm_near_2dly(float* real_result,float* imag_result,const int n,const int combs, const float* real_exp,const float* img_exp,const int* transmit,const int* receive,const int* lookup_ind_tx, const int* lookup_ind_rx,const int tot_pix, const int grid_x, const int grid_y, const int grid_z, const float* lookup_amp_tx,const float* lookup_amp_rx,const float* tt_weight){

	// get pixel's coordinates
    int pix = blockIdx.x*blockDim.x+threadIdx.x;
     
    if (pix<tot_pix){
            //local variable
            float tot_real = 0, tot_imag = 0;

            for(int ii = 0; ii < combs; ii++){
                float real = 0;
                float imag = 0;
                int tx = transmit[ii]-1;
                int rx = receive[ii]-1;
                int t_ind = (tx*grid_x*grid_y*grid_z)+pix;
                int r_ind = (rx*grid_x*grid_y*grid_z)+pix;

                int index = lookup_ind_tx[t_ind] + lookup_ind_rx[r_ind] - 1; 
                float amp_corr = lookup_amp_tx[t_ind]*lookup_amp_rx[r_ind]*tt_weight[ii];
                if(index<0){
                    }
                else if(index>n){
                    }
                else
                    {
                    int set_val = ii*(n)+index;
                    real = real_exp[set_val];
                    real = real*amp_corr;
                    imag = img_exp[set_val];
                    imag = imag*amp_corr;
                    }
                // sum each val
                tot_real += real;
                tot_imag += imag;
					
            }
		
            // store the final value for the pixel
            //result[pix] = sqrt(tot_real*tot_real + tot_imag*tot_imag);
            real_result[pix] = tot_real;
            imag_result[pix] = tot_imag;
    }
}

__global__ void tfm_near_hmc(float* real_result,float* imag_result,const int n,const int combs, const float* real_exp,const float* img_exp,const int* transmit,const int* receive,const int* lookup_ind_tx, const int* lookup_ind_rx,const int tot_pix, const int grid_x, const int grid_y, const int grid_z, const float* lookup_amp_tx,const float* lookup_amp_rx,const float* tt_weight){

	// get pixel's coordinates
    int pix = blockIdx.x*blockDim.x+threadIdx.x;
     
    if (pix<tot_pix){
            //local variable
            float tot_real = 0, tot_imag = 0;

            for(int ii = 0; ii < combs; ii++){
                float real = 0;
                float imag = 0;
                int tx = transmit[ii]-1;
                int rx = receive[ii]-1;
                int t_ind = (tx*grid_x*grid_y*grid_z)+pix;
                int r_ind = (rx*grid_x*grid_y*grid_z)+pix;

                int index1 = lookup_ind_tx[t_ind] + lookup_ind_rx[r_ind]-1; 
                float amp_corr1 = lookup_amp_tx[t_ind]*lookup_amp_rx[r_ind]*tt_weight[ii]/2;
                int index2 = lookup_ind_tx[r_ind] + lookup_ind_rx[t_ind]-1; 
                float amp_corr2 = lookup_amp_tx[r_ind]*lookup_amp_rx[t_ind]*tt_weight[ii]/2;

                if(index1<0){
                    }
                else if(index1>n){
                    }
                else
                    {
                    int set_val1 = ii*(n)+index1;
                    real = real_exp[set_val1]*amp_corr1;
                    imag = img_exp[set_val1]*amp_corr1;
                    // sum each val
                    tot_real += real;
                    tot_imag += imag;
                    }
                real = 0;
                imag = 0;
                if(index2<0){
                    }
                else if(index2>n){
                    }
                else
                    {
                    int set_val2 = ii*(n)+index2;
                    real = real_exp[set_val2]*amp_corr2;
                    imag = img_exp[set_val2]*amp_corr2;
                    // sum each val
                    tot_real += real;
                    tot_imag += imag;
                    }
                
					
            }
		
            // store the final value for the pixel
            //result[pix] = sqrt(tot_real*tot_real + tot_imag*tot_imag);
            real_result[pix] = tot_real;
            imag_result[pix] = tot_imag;
    }
}

__global__ void tfm_linear_norm(float* real_result,float* imag_result,const int n,const int combs, const float* real_exp,const float* img_exp,const int* transmit,const int* receive,const float* lookup_time,const float* time, const int tot_pix, const int grid_x, const int grid_y, const int grid_z, const float* lookup_amp, const float* tt_weight){

	// get pixel's coordinates
    int pix = blockIdx.x*blockDim.x+threadIdx.x;
   
        if (pix<tot_pix){
            //local variable
            float tot_real = 0, tot_imag = 0;
            float dt = time[1]-time[0];

            for(int ii = 0; ii < combs; ii++){
                float real = 0;
                float imag = 0;
                int tx = transmit[ii]-1;
                int rx = receive[ii]-1;
                int t_ind = (tx*grid_x*grid_y*grid_z)+pix;
                int r_ind = (rx*grid_x*grid_y*grid_z)+pix;

                float time_val = lookup_time[t_ind] + lookup_time[r_ind]; 
                float amp_corr = lookup_amp[t_ind]*lookup_amp[r_ind]*tt_weight[ii];
                float time_diff = time_val-time[0];
                if(time_diff<0){
                    }
                else if(time_val > time[n-1]){
                    }
                else
                    {
                    int time_0 = floorf((time_val-time[0])/dt);
                    int set_val = ii*(n)+time_0;
                    float real_y1 = real_exp[set_val];
                    float imag_y1 = img_exp[set_val];
                    float real_y2 = real_exp[set_val+1];
                    float imag_y2 = img_exp[set_val+1];

                    float real_dy = real_y2-real_y1;
                    float imag_dy = imag_y2-imag_y1;
    
                    real = real_y1+real_dy*(time_val-time[time_0])/dt;
                    real = real*amp_corr;
                    imag = imag_y1+imag_dy*(time_val-time[time_0])/dt;
                    imag = imag*amp_corr;
                    }
                // sum each val
                tot_real += real;
                tot_imag += imag;
                    
			}
		
		// store the final value for the pixel
		real_result[pix] = tot_real;
        imag_result[pix] = tot_imag;
    }
}

__global__ void tfm_linear_2dly(float* real_result,float* imag_result,const int n,const int combs, const float* real_exp,const float* img_exp,const int* transmit,const int* receive,const float* lookup_time_tx,const float* lookup_time_rx,const float* time, const int tot_pix, const int grid_x, const int grid_y, const int grid_z, const float* lookup_amp_tx, const float* lookup_amp_rx, const float* tt_weight){

	// get pixel's coordinates
    int pix = blockIdx.x*blockDim.x+threadIdx.x;
   
        if (pix<tot_pix){
            //local variable
            float tot_real = 0, tot_imag = 0;
            float dt = time[1]-time[0];

            for(int ii = 0; ii < combs; ii++){
                float real = 0;
                float imag = 0;
                int tx = transmit[ii]-1;
                int rx = receive[ii]-1;
                int t_ind = (tx*grid_x*grid_y*grid_z)+pix;
                int r_ind = (rx*grid_x*grid_y*grid_z)+pix;

                float time_val = lookup_time_tx[t_ind] + lookup_time_rx[r_ind]; 
                float amp_corr = lookup_amp_tx[t_ind]*lookup_amp_rx[r_ind]*tt_weight[ii];
                float time_diff = time_val-time[0];
                if(time_diff<0){
                    }
                else if(time_val > time[n-1]){
                    }
                else
                    {
                    int time_0 = floorf((time_val-time[0])/dt);
                    int set_val = ii*(n)+time_0;
                    float real_y1 = real_exp[set_val];
                    float imag_y1 = img_exp[set_val];
                    float real_y2 = real_exp[set_val+1];
                    float imag_y2 = img_exp[set_val+1];

                    float real_dy = real_y2-real_y1;
                    float imag_dy = imag_y2-imag_y1;
    
                    real = real_y1+real_dy*(time_val-time[time_0])/dt;
                    real = real*amp_corr;
                    imag = imag_y1+imag_dy*(time_val-time[time_0])/dt;
                    imag = imag*amp_corr;
                    }
                // sum each val
                tot_real += real;
                tot_imag += imag;
                    
			}
		
		// store the final value for the pixel
		real_result[pix] = tot_real;
        imag_result[pix] = tot_imag;
    }
}

__global__ void tfm_linear_hmc(float* real_result,float* imag_result,const int n,const int combs, const float* real_exp,const float* img_exp,const int* transmit,const int* receive,const float* lookup_time_tx,const float* lookup_time_rx,const float* time, const int tot_pix, const int grid_x, const int grid_y, const int grid_z, const float* lookup_amp_tx, const float* lookup_amp_rx, const float* tt_weight){

	// get pixel's coordinates
    int pix = blockIdx.x*blockDim.x+threadIdx.x;
   
        if (pix<tot_pix){
            //local variable
            float tot_real = 0, tot_imag = 0;
            float dt = time[1]-time[0];

            for(int ii = 0; ii < combs; ii++){
                float real = 0;
                float imag = 0;
                int tx = transmit[ii]-1;
                int rx = receive[ii]-1;
                int t_ind = (tx*grid_x*grid_y*grid_z)+pix;
                int r_ind = (rx*grid_x*grid_y*grid_z)+pix;

                float time_val1 = lookup_time_tx[t_ind] + lookup_time_rx[r_ind]; 
                float time_val2 = lookup_time_tx[r_ind] + lookup_time_rx[t_ind]; 
                float amp_corr1 = lookup_amp_tx[t_ind]*lookup_amp_rx[r_ind]*tt_weight[ii]/2;
                float amp_corr2 = lookup_amp_tx[r_ind]*lookup_amp_rx[t_ind]*tt_weight[ii]/2;
                float time_diff1 = time_val1-time[0];
                float time_diff2 = time_val2-time[0];
                if(time_diff1<0){
                    }
                else if(time_val1 > time[n-1]){
                    }
                else
                    {
                    int time_0 = floorf((time_val1-time[0])/dt);
                    int set_val = ii*(n)+time_0;
                    float real_y1 = real_exp[set_val];
                    float imag_y1 = img_exp[set_val];
                    float real_y2 = real_exp[set_val+1];
                    float imag_y2 = img_exp[set_val+1];

                    float real_dy = real_y2-real_y1;
                    float imag_dy = imag_y2-imag_y1;
    
                    real = real_y1+real_dy*(time_val1-time[time_0])/dt;
                    real = real*amp_corr1;
                    imag = imag_y1+imag_dy*(time_val1-time[time_0])/dt;
                    imag = imag*amp_corr1;
                    // sum each val
                    tot_real += real;
                    tot_imag += imag;
                    }
               
                real = 0;
                imag = 0;
                if(time_diff2<0){
                    }
                else if(time_val2 > time[n-1]){
                    }
                else
                    {
                    int time_0 = floorf((time_val2-time[0])/dt);
                    int set_val = ii*(n)+time_0;
                    float real_y1 = real_exp[set_val];
                    float imag_y1 = img_exp[set_val];
                    float real_y2 = real_exp[set_val+1];
                    float imag_y2 = img_exp[set_val+1];

                    float real_dy = real_y2-real_y1;
                    float imag_dy = imag_y2-imag_y1;
    
                    real = real_y1+real_dy*(time_val2-time[time_0])/dt;
                    real = real*amp_corr2;
                    imag = imag_y1+imag_dy*(time_val2-time[time_0])/dt;
                    imag = imag*amp_corr2;
                    // sum each val
                    tot_real += real;
                    tot_imag += imag;
                    }

                    
			}
		
		// store the final value for the pixel
		real_result[pix] = tot_real;
        imag_result[pix] = tot_imag;
    }
}